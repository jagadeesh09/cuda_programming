/*

        Example of using threads and blocks in a CUDA program


                                                    */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


#define N (1024*1024)
#define THREADS_PER_BLOCK 512


__global__ void add(int *a, int *b, int *c){

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];

}

void random_ints(int *a, int n)
{

    int i;
    for (i=0; i < n; ++i){

        a[i] = rand();

    }

}
int main(void) {
    
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    random_ints(b, N);
    random_ints(a, N);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    clock_t t;
    t = clock();

    add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

    t = clock() - t;
    double time_taken = ((double)t)/CLOCKS_PER_SEC;
    printf("Time taken by function is %f seconds\n",time_taken);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_c);
    hipFree(d_c);

    free(a);
    free(b);
    free(c);
    return 0;

}
