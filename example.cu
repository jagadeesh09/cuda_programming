
#include <hip/hip_runtime.h>
#include<stdio.h>


__global__ void mykernel(void) {
}

__global__ void add(int *a, int *b, int *c){

    *c = *a + *b;

}

int main(void) {
    
    mykernel<<<1,1>>>();
    printf("Hello World\n");

    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    a = 2;
    b = 5;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    add<<<1,1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_c);
    hipFree(d_c);
    return 0;

}
